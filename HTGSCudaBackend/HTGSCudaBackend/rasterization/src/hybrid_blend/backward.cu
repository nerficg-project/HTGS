#include "hip/hip_runtime.h"
#include "hybrid_blend/backward.h"
#include "hybrid_blend/kernels/backward.cuh"
#include "hybrid_blend/buffer_utils.h"
#include "hybrid_blend/config.h"
#include "rasterization_utils.h"
#include "utils.h"
#include "hip/hip_vector_types.h"
#include <variant>
#include <utility>

template <typename... Args>
void blend_k_templated(
    const dim3& grid,
    const dim3& block,
    const int K,
    Args&&... kernel_args)
{
    if (K >= 32) htgs::rasterization::hybrid_blend::kernels::backward::blend_cu<32><<<grid, block>>>(std::forward<Args>(kernel_args)...);
    else if (K >= 16) htgs::rasterization::hybrid_blend::kernels::backward::blend_cu<16><<<grid, block>>>(std::forward<Args>(kernel_args)...);
    else if (K >= 8) htgs::rasterization::hybrid_blend::kernels::backward::blend_cu<8><<<grid, block>>>(std::forward<Args>(kernel_args)...);
    else if (K >= 4) htgs::rasterization::hybrid_blend::kernels::backward::blend_cu<4><<<grid, block>>>(std::forward<Args>(kernel_args)...);
    else if (K >= 2) htgs::rasterization::hybrid_blend::kernels::backward::blend_cu<2><<<grid, block>>>(std::forward<Args>(kernel_args)...);
    else htgs::rasterization::hybrid_blend::kernels::backward::blend_cu<1><<<grid, block>>>(std::forward<Args>(kernel_args)...);
}

void htgs::rasterization::hybrid_blend::backward(
    const float* grad_image,
    const float3* positions,
    const float3* scales,
    const float4* rotations,
    const float* opacities,
    const float3* sh_rest,
    const float4* M,
    const float4* VPM,
    const float3* cam_position,
    char* per_primitive_buffers_blob,
    char* per_tile_buffers_blob,
    char* per_instance_buffers_blob,
    char* per_pixel_buffers_blob,
    float3* grad_positions,
    float3* grad_scales,
    float4* grad_rotations,
    float* grad_opacities,
    float3* grad_sh_0,
    float3* grad_sh_rest,
    float* grad_VPMT,
    float* densification_info,
    float* densification_info_helper,
    const int K,
    const int n_primitives,
    const int active_sh_bases,
    const int total_sh_bases,
    const int width,
    const int height,
    const int n_instances,
    const int instance_primitive_indices_selector,
    const bool use_distance_scaling)
{
    if (use_distance_scaling) hipMemcpyToSymbol(HIP_SYMBOL(c_M3), M + 2, sizeof(float4), 0, hipMemcpyDeviceToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_VPM), VPM, 4 * sizeof(float4), 0, hipMemcpyDeviceToDevice);
    hipMemcpyToSymbol(HIP_SYMBOL(c_cam_position), cam_position, sizeof(float3), 0, hipMemcpyDeviceToDevice);

    const dim3 grid(div_round_up(width, config::tile_width), div_round_up(height, config::tile_height), 1);
    const dim3 block(config::tile_width, config::tile_height, 1);
    const int n_tiles = grid.x * grid.y;
    const int end_bit = extract_end_bit(n_tiles);
    const int n_pixels = width * height;

    constexpr bool store_rgba = true, store_rgb_clamp_info = true;
    PerPrimitiveBuffers per_primitive_buffers = PerPrimitiveBuffers::from_blob(per_primitive_buffers_blob, n_primitives, store_rgba, store_rgb_clamp_info);
    PerTileBuffers per_tile_buffers = PerTileBuffers::from_blob(per_tile_buffers_blob, n_tiles);
    PerPixelBuffers per_pixel_buffers = PerPixelBuffers::from_blob(per_pixel_buffers_blob, n_pixels, K);

    std::variant<PerInstanceBuffers<ushort>, PerInstanceBuffers<uint>> buffer_variant;
    if (end_bit <= 16) buffer_variant = PerInstanceBuffers<ushort>::from_blob(per_instance_buffers_blob, n_instances, end_bit);
    else buffer_variant = PerInstanceBuffers<uint>::from_blob(per_instance_buffers_blob, n_instances, end_bit);

    std::visit([&](auto& per_instance_buffers) {
        per_instance_buffers.primitive_indices.selector = instance_primitive_indices_selector;

        blend_k_templated(grid, block, K,
            per_tile_buffers.instance_ranges,
            per_instance_buffers.primitive_indices.Current(),
            per_primitive_buffers.VPMT1,
            per_primitive_buffers.VPMT2,
            per_primitive_buffers.VPMT4,
            per_primitive_buffers.rgba,
            per_pixel_buffers.primitive_indices_core,
            per_pixel_buffers.grad_info_core,
            per_pixel_buffers.grad_info_tail,
            grad_image,
            grad_positions,
            grad_opacities,
            grad_sh_0,
            grad_VPMT,
            densification_info_helper,
            n_primitives,
            width,
            height,
            grid.x
        );
        CHECK_CUDA(config::debug_backward, "blend_backward")

    }, buffer_variant);

    kernels::backward::preprocess_cu<<<div_round_up(n_primitives, config::block_size_preprocess), config::block_size_preprocess>>>(
        positions,
        scales,
        rotations,
        sh_rest,
        per_primitive_buffers.n_touched_tiles,
        per_primitive_buffers.rgb_clamp_info,
        grad_VPMT,
        densification_info_helper,
        grad_positions,
        grad_scales,
        grad_rotations,
        grad_sh_0,
        grad_sh_rest,
        densification_info,
        n_primitives,
        active_sh_bases,
        total_sh_bases,
        use_distance_scaling
    );
    CHECK_CUDA(config::debug_backward, "preprocess_backward")

}
